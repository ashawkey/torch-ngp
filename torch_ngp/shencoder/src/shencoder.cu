#include "hip/hip_runtime.h"
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <algorithm>
#include <stdexcept>

#include <cstdio>


#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")


template <typename T>
__host__ __device__ T div_round_up(T val, T divisor) {
	return (val + divisor - 1) / divisor;
}

template <typename scalar_t>
__global__ void kernel_sh(
    const scalar_t * __restrict__ inputs, 
    scalar_t * outputs, 
    uint32_t B, uint32_t D, uint32_t C,
    scalar_t * dy_dx
) {
	const uint32_t b = threadIdx.x + blockIdx.x * blockDim.x;
	if (b >= B) return;

	const uint32_t C2 = C * C;

	// locate
	inputs += b * D;
	outputs += b * C2;

	scalar_t x = inputs[0], y = inputs[1], z = inputs[2];

	scalar_t xy=x*y, xz=x*z, yz=y*z, x2=x*x, y2=y*y, z2=z*z, xyz=xy*z;
	scalar_t x4=x2*x2, y4=y2*y2, z4=z2*z2;
	scalar_t x6=x4*x2, y6=y4*y2, z6=z4*z2;

	auto write_sh = [&]() {
		outputs[0] = 0.28209479177387814f ;                          // 1/(2*sqrt(pi))
		if (C <= 1) { return; }
		outputs[1] = -0.48860251190291987f*y ;                               // -sqrt(3)*y/(2*sqrt(pi))
		outputs[2] = 0.48860251190291987f*z ;                                // sqrt(3)*z/(2*sqrt(pi))
		outputs[3] = -0.48860251190291987f*x ;                               // -sqrt(3)*x/(2*sqrt(pi))
		if (C <= 2) { return; }
		outputs[4] = 1.0925484305920792f*xy ;                                // sqrt(15)*xy/(2*sqrt(pi))
		outputs[5] = -1.0925484305920792f*yz ;                               // -sqrt(15)*yz/(2*sqrt(pi))
		outputs[6] = 0.94617469575755997f*z2 - 0.31539156525251999f ;                         // sqrt(5)*(3*z2 - 1)/(4*sqrt(pi))
		outputs[7] = -1.0925484305920792f*xz ;                               // -sqrt(15)*xz/(2*sqrt(pi))
		outputs[8] = 0.54627421529603959f*x2 - 0.54627421529603959f*y2 ;                              // sqrt(15)*(x2 - y2)/(4*sqrt(pi))
		if (C <= 3) { return; }
		outputs[9] = 0.59004358992664352f*y*(-3.0f*x2 + y2) ;                         // sqrt(70)*y*(-3*x2 + y2)/(8*sqrt(pi))
		outputs[10] = 2.8906114426405538f*xy*z ;                             // sqrt(105)*xy*z/(2*sqrt(pi))
		outputs[11] = 0.45704579946446572f*y*(1.0f - 5.0f*z2) ;                                // sqrt(42)*y*(1 - 5*z2)/(8*sqrt(pi))
		outputs[12] = 0.3731763325901154f*z*(5.0f*z2 - 3.0f) ;                         // sqrt(7)*z*(5*z2 - 3)/(4*sqrt(pi))
		outputs[13] = 0.45704579946446572f*x*(1.0f - 5.0f*z2) ;                                // sqrt(42)*x*(1 - 5*z2)/(8*sqrt(pi))
		outputs[14] = 1.4453057213202769f*z*(x2 - y2) ;                              // sqrt(105)*z*(x2 - y2)/(4*sqrt(pi))
		outputs[15] = 0.59004358992664352f*x*(-x2 + 3.0f*y2) ;                                // sqrt(70)*x*(-x2 + 3*y2)/(8*sqrt(pi))
		if (C <= 4) { return; }
		outputs[16] = 2.5033429417967046f*xy*(x2 - y2) ;                             // 3*sqrt(35)*xy*(x2 - y2)/(4*sqrt(pi))
		outputs[17] = 1.7701307697799304f*yz*(-3.0f*x2 + y2) ;                                // 3*sqrt(70)*yz*(-3*x2 + y2)/(8*sqrt(pi))
		outputs[18] = 0.94617469575756008f*xy*(7.0f*z2 - 1.0f) ;                               // 3*sqrt(5)*xy*(7*z2 - 1)/(4*sqrt(pi))
		outputs[19] = 0.66904654355728921f*yz*(3.0f - 7.0f*z2) ;                               // 3*sqrt(10)*yz*(3 - 7*z2)/(8*sqrt(pi))
		outputs[20] = -3.1735664074561294f*z2 + 3.7024941420321507f*z4 + 0.31735664074561293f ;                                // 3*(-30*z2 + 35*z4 + 3)/(16*sqrt(pi))
		outputs[21] = 0.66904654355728921f*xz*(3.0f - 7.0f*z2) ;                               // 3*sqrt(10)*xz*(3 - 7*z2)/(8*sqrt(pi))
		outputs[22] = 0.47308734787878004f*(x2 - y2)*(7.0f*z2 - 1.0f) ;                                // 3*sqrt(5)*(x2 - y2)*(7*z2 - 1)/(8*sqrt(pi))
		outputs[23] = 1.7701307697799304f*xz*(-x2 + 3.0f*y2) ;                                // 3*sqrt(70)*xz*(-x2 + 3*y2)/(8*sqrt(pi))
		outputs[24] = -3.7550144126950569f*x2*y2 + 0.62583573544917614f*x4 + 0.62583573544917614f*y4 ;                         // 3*sqrt(35)*(-6*x2*y2 + x4 + y4)/(16*sqrt(pi))
		if (C <= 5) { return; }
		outputs[25] = 0.65638205684017015f*y*(10.0f*x2*y2 - 5.0f*x4 - y4) ;                            // 3*sqrt(154)*y*(10*x2*y2 - 5*x4 - y4)/(32*sqrt(pi))
		outputs[26] = 8.3026492595241645f*xy*z*(x2 - y2) ;                           // 3*sqrt(385)*xy*z*(x2 - y2)/(4*sqrt(pi))
		outputs[27] = -0.48923829943525038f*y*(3.0f*x2 - y2)*(9.0f*z2 - 1.0f) ;                         // -sqrt(770)*y*(3*x2 - y2)*(9*z2 - 1)/(32*sqrt(pi))
		outputs[28] = 4.7935367849733241f*xy*z*(3.0f*z2 - 1.0f) ;                              // sqrt(1155)*xy*z*(3*z2 - 1)/(4*sqrt(pi))
		outputs[29] = 0.45294665119569694f*y*(14.0f*z2 - 21.0f*z4 - 1.0f) ;                             // sqrt(165)*y*(14*z2 - 21*z4 - 1)/(16*sqrt(pi))
		outputs[30] = 0.1169503224534236f*z*(-70.0f*z2 + 63.0f*z4 + 15.0f) ;                            // sqrt(11)*z*(-70*z2 + 63*z4 + 15)/(16*sqrt(pi))
		outputs[31] = 0.45294665119569694f*x*(14.0f*z2 - 21.0f*z4 - 1.0f) ;                             // sqrt(165)*x*(14*z2 - 21*z4 - 1)/(16*sqrt(pi))
		outputs[32] = 2.3967683924866621f*z*(x2 - y2)*(3.0f*z2 - 1.0f) ;                               // sqrt(1155)*z*(x2 - y2)*(3*z2 - 1)/(8*sqrt(pi))
		outputs[33] = -0.48923829943525038f*x*(x2 - 3.0f*y2)*(9.0f*z2 - 1.0f) ;                         // -sqrt(770)*x*(x2 - 3*y2)*(9*z2 - 1)/(32*sqrt(pi))
		outputs[34] = 2.0756623148810411f*z*(-6.0f*x2*y2 + x4 + y4) ;                         // 3*sqrt(385)*z*(-6*x2*y2 + x4 + y4)/(16*sqrt(pi))
		outputs[35] = 0.65638205684017015f*x*(10.0f*x2*y2 - x4 - 5.0f*y4) ;                            // 3*sqrt(154)*x*(10*x2*y2 - x4 - 5*y4)/(32*sqrt(pi))
		if (C <= 6) { return; }
		outputs[36] = 1.3663682103838286f*xy*(-10.0f*x2*y2 + 3.0f*x4 + 3.0f*y4) ;                               // sqrt(6006)*xy*(-10*x2*y2 + 3*x4 + 3*y4)/(32*sqrt(pi))
		outputs[37] = 2.3666191622317521f*yz*(10.0f*x2*y2 - 5.0f*x4 - y4) ;                            // 3*sqrt(2002)*yz*(10*x2*y2 - 5*x4 - y4)/(32*sqrt(pi))
		outputs[38] = 2.0182596029148963f*xy*(x2 - y2)*(11.0f*z2 - 1.0f) ;                             // 3*sqrt(91)*xy*(x2 - y2)*(11*z2 - 1)/(8*sqrt(pi))
		outputs[39] = -0.92120525951492349f*yz*(3.0f*x2 - y2)*(11.0f*z2 - 3.0f) ;                               // -sqrt(2730)*yz*(3*x2 - y2)*(11*z2 - 3)/(32*sqrt(pi))
		outputs[40] = 0.92120525951492349f*xy*(-18.0f*z2 + 33.0f*z4 + 1.0f) ;                           // sqrt(2730)*xy*(-18*z2 + 33*z4 + 1)/(32*sqrt(pi))
		outputs[41] = 0.58262136251873131f*yz*(30.0f*z2 - 33.0f*z4 - 5.0f) ;                            // sqrt(273)*yz*(30*z2 - 33*z4 - 5)/(16*sqrt(pi))
		outputs[42] = 6.6747662381009842f*z2 - 20.024298714302954f*z4 + 14.684485723822165f*z6 - 0.31784601133814211f ;                         // sqrt(13)*(105*z2 - 315*z4 + 231*z6 - 5)/(32*sqrt(pi))
		outputs[43] = 0.58262136251873131f*xz*(30.0f*z2 - 33.0f*z4 - 5.0f) ;                            // sqrt(273)*xz*(30*z2 - 33*z4 - 5)/(16*sqrt(pi))
		outputs[44] = 0.46060262975746175f*(x2 - y2)*(11.0f*z2*(3.0f*z2 - 1.0f) - 7.0f*z2 + 1.0f) ;                               // sqrt(2730)*(x2 - y2)*(11*z2*(3*z2 - 1) - 7*z2 + 1)/(64*sqrt(pi))
		outputs[45] = -0.92120525951492349f*xz*(x2 - 3.0f*y2)*(11.0f*z2 - 3.0f) ;                               // -sqrt(2730)*xz*(x2 - 3*y2)*(11*z2 - 3)/(32*sqrt(pi))
		outputs[46] = 0.50456490072872406f*(11.0f*z2 - 1.0f)*(-6.0f*x2*y2 + x4 + y4) ;                          // 3*sqrt(91)*(11*z2 - 1)*(-6*x2*y2 + x4 + y4)/(32*sqrt(pi))
		outputs[47] = 2.3666191622317521f*xz*(10.0f*x2*y2 - x4 - 5.0f*y4) ;                            // 3*sqrt(2002)*xz*(10*x2*y2 - x4 - 5*y4)/(32*sqrt(pi))
		outputs[48] = 10.247761577878714f*x2*y4 - 10.247761577878714f*x4*y2 + 0.6831841051919143f*x6 - 0.6831841051919143f*y6 ;                         // sqrt(6006)*(15*x2*y4 - 15*x4*y2 + x6 - y6)/(64*sqrt(pi))
		if (C <= 7) { return; }
		outputs[49] = 0.70716273252459627f*y*(-21.0f*x2*y4 + 35.0f*x4*y2 - 7.0f*x6 + y6) ;                              // 3*sqrt(715)*y*(-21*x2*y4 + 35*x4*y2 - 7*x6 + y6)/(64*sqrt(pi))
		outputs[50] = 5.2919213236038001f*xy*z*(-10.0f*x2*y2 + 3.0f*x4 + 3.0f*y4) ;                             // 3*sqrt(10010)*xy*z*(-10*x2*y2 + 3*x4 + 3*y4)/(32*sqrt(pi))
		outputs[51] = -0.51891557872026028f*y*(13.0f*z2 - 1.0f)*(-10.0f*x2*y2 + 5.0f*x4 + y4) ;                          // -3*sqrt(385)*y*(13*z2 - 1)*(-10*x2*y2 + 5*x4 + y4)/(64*sqrt(pi))
		outputs[52] = 4.1513246297620823f*xy*z*(x2 - y2)*(13.0f*z2 - 3.0f) ;                           // 3*sqrt(385)*xy*z*(x2 - y2)*(13*z2 - 3)/(8*sqrt(pi))
		outputs[53] = -0.15645893386229404f*y*(3.0f*x2 - y2)*(13.0f*z2*(11.0f*z2 - 3.0f) - 27.0f*z2 + 3.0f) ;                              // -3*sqrt(35)*y*(3*x2 - y2)*(13*z2*(11*z2 - 3) - 27*z2 + 3)/(64*sqrt(pi))
		outputs[54] = 0.44253269244498261f*xy*z*(-110.0f*z2 + 143.0f*z4 + 15.0f) ;                              // 3*sqrt(70)*xy*z*(-110*z2 + 143*z4 + 15)/(32*sqrt(pi))
		outputs[55] = 0.090331607582517306f*y*(-135.0f*z2 + 495.0f*z4 - 429.0f*z6 + 5.0f) ;                              // sqrt(105)*y*(-135*z2 + 495*z4 - 429*z6 + 5)/(64*sqrt(pi))
		outputs[56] = 0.068284276912004949f*z*(315.0f*z2 - 693.0f*z4 + 429.0f*z6 - 35.0f) ;                              // sqrt(15)*z*(315*z2 - 693*z4 + 429*z6 - 35)/(32*sqrt(pi))
		outputs[57] = 0.090331607582517306f*x*(-135.0f*z2 + 495.0f*z4 - 429.0f*z6 + 5.0f) ;                              // sqrt(105)*x*(-135*z2 + 495*z4 - 429*z6 + 5)/(64*sqrt(pi))
		outputs[58] = 0.07375544874083044f*z*(x2 - y2)*(143.0f*z2*(3.0f*z2 - 1.0f) - 187.0f*z2 + 45.0f) ;                         // sqrt(70)*z*(x2 - y2)*(143*z2*(3*z2 - 1) - 187*z2 + 45)/(64*sqrt(pi))
		outputs[59] = -0.15645893386229404f*x*(x2 - 3.0f*y2)*(13.0f*z2*(11.0f*z2 - 3.0f) - 27.0f*z2 + 3.0f) ;                              // -3*sqrt(35)*x*(x2 - 3*y2)*(13*z2*(11*z2 - 3) - 27*z2 + 3)/(64*sqrt(pi))
		outputs[60] = 1.0378311574405206f*z*(13.0f*z2 - 3.0f)*(-6.0f*x2*y2 + x4 + y4) ;                         // 3*sqrt(385)*z*(13*z2 - 3)*(-6*x2*y2 + x4 + y4)/(32*sqrt(pi))
		outputs[61] = -0.51891557872026028f*x*(13.0f*z2 - 1.0f)*(-10.0f*x2*y2 + x4 + 5.0f*y4) ;                          // -3*sqrt(385)*x*(13*z2 - 1)*(-10*x2*y2 + x4 + 5*y4)/(64*sqrt(pi))
		outputs[62] = 2.6459606618019f*z*(15.0f*x2*y4 - 15.0f*x4*y2 + x6 - y6) ;                               // 3*sqrt(10010)*z*(15*x2*y4 - 15*x4*y2 + x6 - y6)/(64*sqrt(pi))
		outputs[63] = 0.70716273252459627f*x*(-35.0f*x2*y4 + 21.0f*x4*y2 - x6 + 7.0f*y6) ;                              // 3*sqrt(715)*x*(-35*x2*y4 + 21*x4*y2 - x6 + 7*y6)/(64*sqrt(pi))
	};

	write_sh();

	if (dy_dx) {
		scalar_t *dx = dy_dx + b * D * C2;
		scalar_t *dy = dx + C2;
		scalar_t *dz = dy + C2;

		auto write_sh_dx = [&]() {
			dx[0] = 0.0f ;                             // 0
			if (C <= 1) { return; }
			dx[1] = 0.0f ;                             // 0
			dx[2] = 0.0f ;                             // 0
			dx[3] = -0.48860251190291992f ;                          // -sqrt(3)/(2*sqrt(pi))
			if (C <= 2) { return; }
			dx[4] = 1.0925484305920792f*y ;                          // sqrt(15)*y/(2*sqrt(pi))
			dx[5] = 0.0f ;                             // 0
			dx[6] = 0.0f ;                             // 0
			dx[7] = -1.0925484305920792f*z ;                         // -sqrt(15)*z/(2*sqrt(pi))
			dx[8] = 1.0925484305920792f*x ;                          // sqrt(15)*x/(2*sqrt(pi))
			if (C <= 3) { return; }
			dx[9] = -3.5402615395598609f*xy ;                                // -3*sqrt(70)*xy/(4*sqrt(pi))
			dx[10] = 2.8906114426405538f*yz ;                                // sqrt(105)*yz/(2*sqrt(pi))
			dx[11] = 0.0f ;                            // 0
			dx[12] = 0.0f ;                            // 0
			dx[13] = 0.45704579946446572f - 2.2852289973223288f*z2 ;                          // sqrt(42)*(1 - 5*z2)/(8*sqrt(pi))
			dx[14] = 2.8906114426405538f*xz ;                                // sqrt(105)*xz/(2*sqrt(pi))
			dx[15] = -1.7701307697799304f*x2 + 1.7701307697799304f*y2 ;                               // 3*sqrt(70)*(-x2 + y2)/(8*sqrt(pi))
			if (C <= 4) { return; }
			dx[16] = 2.5033429417967046f*y*(3.0f*x2 - y2) ;                           // 3*sqrt(35)*y*(3*x2 - y2)/(4*sqrt(pi))
			dx[17] = -10.620784618679583f*xy*z ;                             // -9*sqrt(70)*xy*z/(4*sqrt(pi))
			dx[18] = 0.94617469575756008f*y*(7.0f*z2 - 1.0f) ;                         // 3*sqrt(5)*y*(7*z2 - 1)/(4*sqrt(pi))
			dx[19] = 0.0f ;                            // 0
			dx[20] = 0.0f ;                            // 0
			dx[21] = 0.66904654355728921f*z*(3.0f - 7.0f*z2) ;                         // 3*sqrt(10)*z*(3 - 7*z2)/(8*sqrt(pi))
			dx[22] = 0.94617469575756008f*x*(7.0f*z2 - 1.0f) ;                         // 3*sqrt(5)*x*(7*z2 - 1)/(4*sqrt(pi))
			dx[23] = 5.3103923093397913f*z*(-x2 + y2) ;                              // 9*sqrt(70)*z*(-x2 + y2)/(8*sqrt(pi))
			dx[24] = 2.5033429417967046f*x*(x2 - 3.0f*y2) ;                           // 3*sqrt(35)*x*(x2 - 3*y2)/(4*sqrt(pi))
			if (C <= 5) { return; }
			dx[25] = 13.127641136803401f*xy*(-x2 + y2) ;                             // 15*sqrt(154)*xy*(-x2 + y2)/(8*sqrt(pi))
			dx[26] = 8.3026492595241645f*yz*(3.0f*x2 - y2) ;                          // 3*sqrt(385)*yz*(3*x2 - y2)/(4*sqrt(pi))
			dx[27] = 2.9354297966115022f*xy*(1.0f - 9.0f*z2) ;                         // 3*sqrt(770)*xy*(1 - 9*z2)/(16*sqrt(pi))
			dx[28] = 4.7935367849733241f*yz*(3.0f*z2 - 1.0f) ;                         // sqrt(1155)*yz*(3*z2 - 1)/(4*sqrt(pi))
			dx[29] = 0.0f ;                            // 0
			dx[30] = 0.0f ;                            // 0
			dx[31] = 6.3412531167397574f*z2 - 9.5118796751096362f*z4 - 0.45294665119569694f ;                          // sqrt(165)*(14*z2 - 21*z4 - 1)/(16*sqrt(pi))
			dx[32] = 4.7935367849733241f*xz*(3.0f*z2 - 1.0f) ;                         // sqrt(1155)*xz*(3*z2 - 1)/(4*sqrt(pi))
			dx[33] = -13.209434084751759f*x2*z2 + 1.4677148983057511f*x2 + 13.209434084751759f*y2*z2 - 1.4677148983057511f*y2 ;                         // 3*sqrt(770)*(-9*x2*z2 + x2 + 9*y2*z2 - y2)/(32*sqrt(pi))
			dx[34] = 8.3026492595241645f*xz*(x2 - 3.0f*y2) ;                          // 3*sqrt(385)*xz*(x2 - 3*y2)/(4*sqrt(pi))
			dx[35] = 19.6914617052051f*x2*y2 - 3.2819102842008503f*x4 - 3.2819102842008503f*y4 ;                               // 15*sqrt(154)*(6*x2*y2 - x4 - y4)/(32*sqrt(pi))
			if (C <= 6) { return; }
			dx[36] = 4.0991046311514854f*y*(-10.0f*x2*y2 + 5.0f*x4 + y4) ;                             // 3*sqrt(6006)*y*(-10*x2*y2 + 5*x4 + y4)/(32*sqrt(pi))
			dx[37] = 47.332383244635047f*xy*z*(-x2 + y2) ;                           // 15*sqrt(2002)*xy*z*(-x2 + y2)/(8*sqrt(pi))
			dx[38] = 2.0182596029148963f*y*(3.0f*x2 - y2)*(11.0f*z2 - 1.0f) ;                           // 3*sqrt(91)*y*(3*x2 - y2)*(11*z2 - 1)/(8*sqrt(pi))
			dx[39] = 5.5272315570895412f*xy*z*(3.0f - 11.0f*z2) ;                              // 3*sqrt(2730)*xy*z*(3 - 11*z2)/(16*sqrt(pi))
			dx[40] = 0.92120525951492349f*y*(-18.0f*z2 + 33.0f*z4 + 1.0f) ;                             // sqrt(2730)*y*(-18*z2 + 33*z4 + 1)/(32*sqrt(pi))
			dx[41] = 0.0f ;                            // 0
			dx[42] = 0.0f ;                            // 0
			dx[43] = 0.58262136251873131f*z*(30.0f*z2 - 33.0f*z4 - 5.0f) ;                              // sqrt(273)*z*(30*z2 - 33*z4 - 5)/(16*sqrt(pi))
			dx[44] = 0.92120525951492349f*x*(-18.0f*z2 + 33.0f*z4 + 1.0f) ;                             // sqrt(2730)*x*(-18*z2 + 33*z4 + 1)/(32*sqrt(pi))
			dx[45] = -2.7636157785447706f*z*(x2 - y2)*(11.0f*z2 - 3.0f) ;                              // -3*sqrt(2730)*z*(x2 - y2)*(11*z2 - 3)/(32*sqrt(pi))
			dx[46] = 2.0182596029148963f*x*(x2 - 3.0f*y2)*(11.0f*z2 - 1.0f) ;                           // 3*sqrt(91)*x*(x2 - 3*y2)*(11*z2 - 1)/(8*sqrt(pi))
			dx[47] = 11.833095811158762f*z*(6.0f*x2*y2 - x4 - y4) ;                           // 15*sqrt(2002)*z*(6*x2*y2 - x4 - y4)/(32*sqrt(pi))
			dx[48] = 4.0991046311514854f*x*(-10.0f*x2*y2 + x4 + 5.0f*y4) ;                             // 3*sqrt(6006)*x*(-10*x2*y2 + x4 + 5*y4)/(32*sqrt(pi))
			if (C <= 7) { return; }
			dx[49] = 9.9002782553443485f*xy*(10.0f*x2*y2 - 3.0f*x4 - 3.0f*y4) ;                         // 21*sqrt(715)*xy*(10*x2*y2 - 3*x4 - 3*y4)/(32*sqrt(pi))
			dx[50] = 15.875763970811402f*yz*(-10.0f*x2*y2 + 5.0f*x4 + y4) ;                            // 9*sqrt(10010)*yz*(-10*x2*y2 + 5*x4 + y4)/(32*sqrt(pi))
			dx[51] = -10.378311574405206f*xy*(x2 - y2)*(13.0f*z2 - 1.0f) ;                             // -15*sqrt(385)*xy*(x2 - y2)*(13*z2 - 1)/(16*sqrt(pi))
			dx[52] = 4.1513246297620823f*yz*(3.0f*x2 - y2)*(13.0f*z2 - 3.0f) ;                          // 3*sqrt(385)*yz*(3*x2 - y2)*(13*z2 - 3)/(8*sqrt(pi))
			dx[53] = 0.93875360317376422f*xy*(66.0f*z2 - 143.0f*z4 - 3.0f) ;                            // 9*sqrt(35)*xy*(66*z2 - 143*z4 - 3)/(32*sqrt(pi))
			dx[54] = 0.44253269244498261f*yz*(-110.0f*z2 + 143.0f*z4 + 15.0f) ;                         // 3*sqrt(70)*yz*(-110*z2 + 143*z4 + 15)/(32*sqrt(pi))
			dx[55] = 0.0f ;                            // 0
			dx[56] = 0.0f ;                            // 0
			dx[57] = -12.194767023639836f*z2 + 44.714145753346067f*z4 - 38.752259652899923f*z6 + 0.45165803791258652f ;                         // sqrt(105)*(-135*z2 + 495*z4 - 429*z6 + 5)/(64*sqrt(pi))
			dx[58] = 0.44253269244498261f*xz*(-110.0f*z2 + 143.0f*z4 + 15.0f) ;                         // 3*sqrt(70)*xz*(-110*z2 + 143*z4 + 15)/(32*sqrt(pi))
			dx[59] = 30.97886890473422f*x2*z2 - 67.120882626924143f*x2*z4 - 1.4081304047606462f*x2 - 30.97886890473422f*y2*z2 + 67.120882626924143f*y2*z4 + 1.4081304047606462f*y2 ;                              // 9*sqrt(35)*(66*x2*z2 - 143*x2*z4 - 3*x2 - 66*y2*z2 + 143*y2*z4 + 3*y2)/(64*sqrt(pi))
			dx[60] = 4.1513246297620823f*xz*(x2 - 3.0f*y2)*(13.0f*z2 - 3.0f) ;                          // 3*sqrt(385)*xz*(x2 - 3*y2)*(13*z2 - 3)/(8*sqrt(pi))
			dx[61] = -0.51891557872026028f*(13.0f*z2 - 1.0f)*(-10.0f*x2*y2 + 4.0f*x2*(x2 - 5.0f*y2) + x4 + 5.0f*y4) ;                              // -3*sqrt(385)*(13*z2 - 1)*(-10*x2*y2 + 4*x2*(x2 - 5*y2) + x4 + 5*y4)/(64*sqrt(pi))
			dx[62] = 15.875763970811402f*xz*(-10.0f*x2*y2 + x4 + 5.0f*y4) ;                            // 9*sqrt(10010)*xz*(-10*x2*y2 + x4 + 5*y4)/(32*sqrt(pi))
			dx[63] = -74.252086915082614f*x2*y4 + 74.252086915082614f*x4*y2 - 4.9501391276721742f*x6 + 4.9501391276721742f*y6 ;                         // 21*sqrt(715)*(-15*x2*y4 + 15*x4*y2 - x6 + y6)/(64*sqrt(pi))
		};

		auto write_sh_dy = [&]() {
			dy[0] = 0.0f ;                             // 0
			if (C <= 1) { return; }
			dy[1] = -0.48860251190291992f ;                          // -sqrt(3)/(2*sqrt(pi))
			dy[2] = 0.0f ;                             // 0
			dy[3] = 0.0f ;                             // 0
			if (C <= 2) { return; }
			dy[4] = 1.0925484305920792f*x ;                          // sqrt(15)*x/(2*sqrt(pi))
			dy[5] = -1.0925484305920792f*z ;                         // -sqrt(15)*z/(2*sqrt(pi))
			dy[6] = 0.0f ;                             // 0
			dy[7] = 0.0f ;                             // 0
			dy[8] = -1.0925484305920792f*y ;                         // -sqrt(15)*y/(2*sqrt(pi))
			if (C <= 3) { return; }
			dy[9] = -1.7701307697799304f*x2 + 1.7701307697799304f*y2 ;                                // 3*sqrt(70)*(-x2 + y2)/(8*sqrt(pi))
			dy[10] = 2.8906114426405538f*xz ;                                // sqrt(105)*xz/(2*sqrt(pi))
			dy[11] = 0.45704579946446572f - 2.2852289973223288f*z2 ;                          // sqrt(42)*(1 - 5*z2)/(8*sqrt(pi))
			dy[12] = 0.0f ;                            // 0
			dy[13] = 0.0f ;                            // 0
			dy[14] = -2.8906114426405538f*yz ;                               // -sqrt(105)*yz/(2*sqrt(pi))
			dy[15] = 3.5402615395598609f*xy ;                                // 3*sqrt(70)*xy/(4*sqrt(pi))
			if (C <= 4) { return; }
			dy[16] = 2.5033429417967046f*x*(x2 - 3.0f*y2) ;                           // 3*sqrt(35)*x*(x2 - 3*y2)/(4*sqrt(pi))
			dy[17] = 5.3103923093397913f*z*(-x2 + y2) ;                              // 9*sqrt(70)*z*(-x2 + y2)/(8*sqrt(pi))
			dy[18] = 0.94617469575756008f*x*(7.0f*z2 - 1.0f) ;                         // 3*sqrt(5)*x*(7*z2 - 1)/(4*sqrt(pi))
			dy[19] = 0.66904654355728921f*z*(3.0f - 7.0f*z2) ;                         // 3*sqrt(10)*z*(3 - 7*z2)/(8*sqrt(pi))
			dy[20] = 0.0f ;                            // 0
			dy[21] = 0.0f ;                            // 0
			dy[22] = 0.94617469575756008f*y*(1.0f - 7.0f*z2) ;                         // 3*sqrt(5)*y*(1 - 7*z2)/(4*sqrt(pi))
			dy[23] = 10.620784618679583f*xy*z ;                              // 9*sqrt(70)*xy*z/(4*sqrt(pi))
			dy[24] = 2.5033429417967046f*y*(-3.0f*x2 + y2) ;                          // 3*sqrt(35)*y*(-3*x2 + y2)/(4*sqrt(pi))
			if (C <= 5) { return; }
			dy[25] = 19.6914617052051f*x2*y2 - 3.2819102842008503f*x4 - 3.2819102842008503f*y4 ;                               // 15*sqrt(154)*(6*x2*y2 - x4 - y4)/(32*sqrt(pi))
			dy[26] = 8.3026492595241645f*xz*(x2 - 3.0f*y2) ;                          // 3*sqrt(385)*xz*(x2 - 3*y2)/(4*sqrt(pi))
			dy[27] = -1.4677148983057511f*(x2 - y2)*(9.0f*z2 - 1.0f) ;                         // -3*sqrt(770)*(x2 - y2)*(9*z2 - 1)/(32*sqrt(pi))
			dy[28] = 4.7935367849733241f*xz*(3.0f*z2 - 1.0f) ;                         // sqrt(1155)*xz*(3*z2 - 1)/(4*sqrt(pi))
			dy[29] = 6.3412531167397574f*z2 - 9.5118796751096362f*z4 - 0.45294665119569694f ;                          // sqrt(165)*(14*z2 - 21*z4 - 1)/(16*sqrt(pi))
			dy[30] = 0.0f ;                            // 0
			dy[31] = 0.0f ;                            // 0
			dy[32] = 4.7935367849733241f*yz*(1.0f - 3.0f*z2) ;                         // sqrt(1155)*yz*(1 - 3*z2)/(4*sqrt(pi))
			dy[33] = 2.9354297966115022f*xy*(9.0f*z2 - 1.0f) ;                         // 3*sqrt(770)*xy*(9*z2 - 1)/(16*sqrt(pi))
			dy[34] = 8.3026492595241645f*yz*(-3.0f*x2 + y2) ;                         // 3*sqrt(385)*yz*(-3*x2 + y2)/(4*sqrt(pi))
			dy[35] = 13.127641136803401f*xy*(x2 - y2) ;                              // 15*sqrt(154)*xy*(x2 - y2)/(8*sqrt(pi))
			if (C <= 6) { return; }
			dy[36] = 4.0991046311514854f*x*(-10.0f*x2*y2 + x4 + 5.0f*y4) ;                             // 3*sqrt(6006)*x*(-10*x2*y2 + x4 + 5*y4)/(32*sqrt(pi))
			dy[37] = 11.833095811158762f*z*(6.0f*x2*y2 - x4 - y4) ;                           // 15*sqrt(2002)*z*(6*x2*y2 - x4 - y4)/(32*sqrt(pi))
			dy[38] = 2.0182596029148963f*x*(x2 - 3.0f*y2)*(11.0f*z2 - 1.0f) ;                           // 3*sqrt(91)*x*(x2 - 3*y2)*(11*z2 - 1)/(8*sqrt(pi))
			dy[39] = -2.7636157785447706f*z*(x2 - y2)*(11.0f*z2 - 3.0f) ;                              // -3*sqrt(2730)*z*(x2 - y2)*(11*z2 - 3)/(32*sqrt(pi))
			dy[40] = 0.92120525951492349f*x*(-18.0f*z2 + 33.0f*z4 + 1.0f) ;                             // sqrt(2730)*x*(-18*z2 + 33*z4 + 1)/(32*sqrt(pi))
			dy[41] = 0.58262136251873131f*z*(30.0f*z2 - 33.0f*z4 - 5.0f) ;                              // sqrt(273)*z*(30*z2 - 33*z4 - 5)/(16*sqrt(pi))
			dy[42] = 0.0f ;                            // 0
			dy[43] = 0.0f ;                            // 0
			dy[44] = 0.92120525951492349f*y*(18.0f*z2 - 33.0f*z4 - 1.0f) ;                              // sqrt(2730)*y*(18*z2 - 33*z4 - 1)/(32*sqrt(pi))
			dy[45] = 5.5272315570895412f*xy*z*(11.0f*z2 - 3.0f) ;                              // 3*sqrt(2730)*xy*z*(11*z2 - 3)/(16*sqrt(pi))
			dy[46] = -2.0182596029148963f*y*(3.0f*x2 - y2)*(11.0f*z2 - 1.0f) ;                          // -3*sqrt(91)*y*(3*x2 - y2)*(11*z2 - 1)/(8*sqrt(pi))
			dy[47] = 47.332383244635047f*xy*z*(x2 - y2) ;                            // 15*sqrt(2002)*xy*z*(x2 - y2)/(8*sqrt(pi))
			dy[48] = 4.0991046311514854f*y*(10.0f*x2*y2 - 5.0f*x4 - y4) ;                              // 3*sqrt(6006)*y*(10*x2*y2 - 5*x4 - y4)/(32*sqrt(pi))
			if (C <= 7) { return; }
			dy[49] = -74.252086915082614f*x2*y4 + 74.252086915082614f*x4*y2 - 4.9501391276721742f*x6 + 4.9501391276721742f*y6 ;                         // 21*sqrt(715)*(-15*x2*y4 + 15*x4*y2 - x6 + y6)/(64*sqrt(pi))
			dy[50] = 15.875763970811402f*xz*(-10.0f*x2*y2 + x4 + 5.0f*y4) ;                            // 9*sqrt(10010)*xz*(-10*x2*y2 + x4 + 5*y4)/(32*sqrt(pi))
			dy[51] = 0.51891557872026028f*(13.0f*z2 - 1.0f)*(10.0f*x2*y2 - 5.0f*x4 + 4.0f*y2*(5.0f*x2 - y2) - y4) ;                                // 3*sqrt(385)*(13*z2 - 1)*(10*x2*y2 - 5*x4 + 4*y2*(5*x2 - y2) - y4)/(64*sqrt(pi))
			dy[52] = 4.1513246297620823f*xz*(x2 - 3.0f*y2)*(13.0f*z2 - 3.0f) ;                          // 3*sqrt(385)*xz*(x2 - 3*y2)*(13*z2 - 3)/(8*sqrt(pi))
			dy[53] = -0.46937680158688211f*(x2 - y2)*(13.0f*z2*(11.0f*z2 - 3.0f) - 27.0f*z2 + 3.0f) ;                             // -9*sqrt(35)*(x2 - y2)*(13*z2*(11*z2 - 3) - 27*z2 + 3)/(64*sqrt(pi))
			dy[54] = 0.44253269244498261f*xz*(-110.0f*z2 + 143.0f*z4 + 15.0f) ;                         // 3*sqrt(70)*xz*(-110*z2 + 143*z4 + 15)/(32*sqrt(pi))
			dy[55] = -12.194767023639836f*z2 + 44.714145753346067f*z4 - 38.752259652899923f*z6 + 0.45165803791258652f ;                         // sqrt(105)*(-135*z2 + 495*z4 - 429*z6 + 5)/(64*sqrt(pi))
			dy[56] = 0.0f ;                            // 0
			dy[57] = 0.0f ;                            // 0
			dy[58] = 0.44253269244498261f*yz*(110.0f*z2 - 143.0f*z4 - 15.0f) ;                          // 3*sqrt(70)*yz*(110*z2 - 143*z4 - 15)/(32*sqrt(pi))
			dy[59] = 0.93875360317376422f*xy*(-66.0f*z2 + 143.0f*z4 + 3.0f) ;                           // 9*sqrt(35)*xy*(-66*z2 + 143*z4 + 3)/(32*sqrt(pi))
			dy[60] = -4.1513246297620823f*yz*(3.0f*x2 - y2)*(13.0f*z2 - 3.0f) ;                         // -3*sqrt(385)*yz*(3*x2 - y2)*(13*z2 - 3)/(8*sqrt(pi))
			dy[61] = 10.378311574405206f*xy*(x2 - y2)*(13.0f*z2 - 1.0f) ;                              // 15*sqrt(385)*xy*(x2 - y2)*(13*z2 - 1)/(16*sqrt(pi))
			dy[62] = 15.875763970811402f*yz*(10.0f*x2*y2 - 5.0f*x4 - y4) ;                             // 9*sqrt(10010)*yz*(10*x2*y2 - 5*x4 - y4)/(32*sqrt(pi))
			dy[63] = 9.9002782553443485f*xy*(-10.0f*x2*y2 + 3.0f*x4 + 3.0f*y4) ;                                // 21*sqrt(715)*xy*(-10*x2*y2 + 3*x4 + 3*y4)/(32*sqrt(pi))
		};

		auto write_sh_dz = [&]() {
			dz[0] = 0.0f ;                             // 0
			if (C <= 1) { return; }
			dz[1] = 0.0f ;                             // 0
			dz[2] = 0.48860251190291992f ;                           // sqrt(3)/(2*sqrt(pi))
			dz[3] = 0.0f ;                             // 0
			if (C <= 2) { return; }
			dz[4] = 0.0f ;                             // 0
			dz[5] = -1.0925484305920792f*y ;                         // -sqrt(15)*y/(2*sqrt(pi))
			dz[6] = 1.8923493915151202f*z ;                          // 3*sqrt(5)*z/(2*sqrt(pi))
			dz[7] = -1.0925484305920792f*x ;                         // -sqrt(15)*x/(2*sqrt(pi))
			dz[8] = 0.0f ;                             // 0
			if (C <= 3) { return; }
			dz[9] = 0.0f ;                             // 0
			dz[10] = 2.8906114426405538f*xy ;                                // sqrt(105)*xy/(2*sqrt(pi))
			dz[11] = -4.5704579946446566f*yz ;                               // -5*sqrt(42)*yz/(4*sqrt(pi))
			dz[12] = 5.597644988851731f*z2 - 1.1195289977703462f ;                            // 3*sqrt(7)*(5*z2 - 1)/(4*sqrt(pi))
			dz[13] = -4.5704579946446566f*xz ;                               // -5*sqrt(42)*xz/(4*sqrt(pi))
			dz[14] = 1.4453057213202769f*x2 - 1.4453057213202769f*y2 ;                                // sqrt(105)*(x2 - y2)/(4*sqrt(pi))
			dz[15] = 0.0f ;                            // 0
			if (C <= 4) { return; }
			dz[16] = 0.0f ;                            // 0
			dz[17] = 1.7701307697799304f*y*(-3.0f*x2 + y2) ;                          // 3*sqrt(70)*y*(-3*x2 + y2)/(8*sqrt(pi))
			dz[18] = 13.246445740605839f*xy*z ;                              // 21*sqrt(5)*xy*z/(2*sqrt(pi))
			dz[19] = 2.0071396306718676f*y*(1.0f - 7.0f*z2) ;                          // 9*sqrt(10)*y*(1 - 7*z2)/(8*sqrt(pi))
			dz[20] = 14.809976568128603f*pow(z, 3) - 6.3471328149122579f*z ;                          // (105*z**3 - 45*z)/(4*sqrt(pi))
			dz[21] = 2.0071396306718676f*x*(1.0f - 7.0f*z2) ;                          // 9*sqrt(10)*x*(1 - 7*z2)/(8*sqrt(pi))
			dz[22] = 6.6232228703029197f*z*(x2 - y2) ;                               // 21*sqrt(5)*z*(x2 - y2)/(4*sqrt(pi))
			dz[23] = 1.7701307697799304f*x*(-x2 + 3.0f*y2) ;                          // 3*sqrt(70)*x*(-x2 + 3*y2)/(8*sqrt(pi))
			dz[24] = 0.0f ;                            // 0
			if (C <= 5) { return; }
			dz[25] = 0.0f ;                            // 0
			dz[26] = 8.3026492595241645f*xy*(x2 - y2) ;                              // 3*sqrt(385)*xy*(x2 - y2)/(4*sqrt(pi))
			dz[27] = 8.8062893898345074f*yz*(-3.0f*x2 + y2) ;                         // 9*sqrt(770)*yz*(-3*x2 + y2)/(16*sqrt(pi))
			dz[28] = 4.7935367849733241f*xy*(9.0f*z2 - 1.0f) ;                         // sqrt(1155)*xy*(9*z2 - 1)/(4*sqrt(pi))
			dz[29] = 12.682506233479513f*yz*(1.0f - 3.0f*z2) ;                         // 7*sqrt(165)*yz*(1 - 3*z2)/(4*sqrt(pi))
			dz[30] = -24.559567715218954f*z2 + 36.839351572828434f*z4 + 1.754254836801354f ;                           // 15*sqrt(11)*(-14*z2 + 21*z4 + 1)/(16*sqrt(pi))
			dz[31] = 12.682506233479513f*xz*(1.0f - 3.0f*z2) ;                         // 7*sqrt(165)*xz*(1 - 3*z2)/(4*sqrt(pi))
			dz[32] = 2.3967683924866621f*(x2 - y2)*(9.0f*z2 - 1.0f) ;                          // sqrt(1155)*(x2 - y2)*(9*z2 - 1)/(8*sqrt(pi))
			dz[33] = 8.8062893898345074f*xz*(-x2 + 3.0f*y2) ;                         // 9*sqrt(770)*xz*(-x2 + 3*y2)/(16*sqrt(pi))
			dz[34] = -12.453973889286246f*x2*y2 + 2.0756623148810411f*x4 + 2.0756623148810411f*y4 ;                            // 3*sqrt(385)*(-6*x2*y2 + x4 + y4)/(16*sqrt(pi))
			dz[35] = 0.0f ;                            // 0
			if (C <= 6) { return; }
			dz[36] = 0.0f ;                            // 0
			dz[37] = 2.3666191622317521f*y*(10.0f*x2*y2 - 5.0f*x4 - y4) ;                              // 3*sqrt(2002)*y*(10*x2*y2 - 5*x4 - y4)/(32*sqrt(pi))
			dz[38] = 44.401711264127719f*xy*z*(x2 - y2) ;                            // 33*sqrt(91)*xy*z*(x2 - y2)/(4*sqrt(pi))
			dz[39] = -2.7636157785447706f*y*(3.0f*x2 - y2)*(11.0f*z2 - 1.0f) ;                          // -3*sqrt(2730)*y*(3*x2 - y2)*(11*z2 - 1)/(32*sqrt(pi))
			dz[40] = 11.054463114179082f*xy*z*(11.0f*z2 - 3.0f) ;                              // 3*sqrt(2730)*xy*z*(11*z2 - 3)/(8*sqrt(pi))
			dz[41] = 2.9131068125936568f*y*(18.0f*z2 - 33.0f*z4 - 1.0f) ;                               // 5*sqrt(273)*y*(18*z2 - 33*z4 - 1)/(16*sqrt(pi))
			dz[42] = 2.6699064952403937f*z*(-30.0f*z2 + 33.0f*z4 + 5.0f) ;                              // 21*sqrt(13)*z*(-30*z2 + 33*z4 + 5)/(16*sqrt(pi))
			dz[43] = 2.9131068125936568f*x*(18.0f*z2 - 33.0f*z4 - 1.0f) ;                               // 5*sqrt(273)*x*(18*z2 - 33*z4 - 1)/(16*sqrt(pi))
			dz[44] = 5.5272315570895412f*z*(x2 - y2)*(11.0f*z2 - 3.0f) ;                               // 3*sqrt(2730)*z*(x2 - y2)*(11*z2 - 3)/(16*sqrt(pi))
			dz[45] = -2.7636157785447706f*x*(x2 - 3.0f*y2)*(11.0f*z2 - 1.0f) ;                          // -3*sqrt(2730)*x*(x2 - 3*y2)*(11*z2 - 1)/(32*sqrt(pi))
			dz[46] = 11.10042781603193f*z*(-6.0f*x2*y2 + x4 + y4) ;                           // 33*sqrt(91)*z*(-6*x2*y2 + x4 + y4)/(16*sqrt(pi))
			dz[47] = 2.3666191622317521f*x*(10.0f*x2*y2 - x4 - 5.0f*y4) ;                              // 3*sqrt(2002)*x*(10*x2*y2 - x4 - 5*y4)/(32*sqrt(pi))
			dz[48] = 0.0f ;                            // 0
			if (C <= 7) { return; }
			dz[49] = 0.0f ;                            // 0
			dz[50] = 5.2919213236038001f*xy*(-10.0f*x2*y2 + 3.0f*x4 + 3.0f*y4) ;                                // 3*sqrt(10010)*xy*(-10*x2*y2 + 3*x4 + 3*y4)/(32*sqrt(pi))
			dz[51] = 13.491805046726766f*yz*(10.0f*x2*y2 - 5.0f*x4 - y4) ;                             // 39*sqrt(385)*yz*(10*x2*y2 - 5*x4 - y4)/(32*sqrt(pi))
			dz[52] = 12.453973889286248f*xy*(x2 - y2)*(13.0f*z2 - 1.0f) ;                              // 9*sqrt(385)*xy*(x2 - y2)*(13*z2 - 1)/(8*sqrt(pi))
			dz[53] = -6.8841930899409371f*yz*(3.0f*x2 - y2)*(13.0f*z2 - 3.0f) ;                         // -33*sqrt(35)*yz*(3*x2 - y2)*(13*z2 - 3)/(16*sqrt(pi))
			dz[54] = 2.2126634622249131f*xy*(-66.0f*z2 + 143.0f*z4 + 3.0f) ;                            // 15*sqrt(70)*xy*(-66*z2 + 143*z4 + 3)/(32*sqrt(pi))
			dz[55] = 1.6259689364853116f*yz*(110.0f*z2 - 143.0f*z4 - 15.0f) ;                           // 9*sqrt(105)*yz*(110*z2 - 143*z4 - 15)/(32*sqrt(pi))
			dz[56] = 64.528641681844675f*z2 - 236.60501950009714f*z4 + 205.05768356675085f*z6 - 2.3899496919201733f ;                           // 7*sqrt(15)*(135*z2 - 495*z4 + 429*z6 - 5)/(32*sqrt(pi))
			dz[57] = 1.6259689364853116f*xz*(110.0f*z2 - 143.0f*z4 - 15.0f) ;                           // 9*sqrt(105)*xz*(110*z2 - 143*z4 - 15)/(32*sqrt(pi))
			dz[58] = 0.07375544874083044f*(x2 - y2)*(143.0f*z2*(3.0f*z2 - 1.0f) + 132.0f*z2*(13.0f*z2 - 5.0f) - 187.0f*z2 + 45.0f) ;                         // sqrt(70)*(x2 - y2)*(143*z2*(3*z2 - 1) + 132*z2*(13*z2 - 5) - 187*z2 + 45)/(64*sqrt(pi))
			dz[59] = -6.8841930899409371f*xz*(x2 - 3.0f*y2)*(13.0f*z2 - 3.0f) ;                         // -33*sqrt(35)*xz*(x2 - 3*y2)*(13*z2 - 3)/(16*sqrt(pi))
			dz[60] = 3.1134934723215619f*(13.0f*z2 - 1.0f)*(-6.0f*x2*y2 + x4 + y4) ;                            // 9*sqrt(385)*(13*z2 - 1)*(-6*x2*y2 + x4 + y4)/(32*sqrt(pi))
			dz[61] = 13.491805046726766f*xz*(10.0f*x2*y2 - x4 - 5.0f*y4) ;                             // 39*sqrt(385)*xz*(10*x2*y2 - x4 - 5*y4)/(32*sqrt(pi))
			dz[62] = 39.6894099270285f*x2*y4 - 39.6894099270285f*x4*y2 + 2.6459606618019f*x6 - 2.6459606618019f*y6 ;                            // 3*sqrt(10010)*(15*x2*y4 - 15*x4*y2 + x6 - y6)/(64*sqrt(pi))
			dz[63] = 0.0f ;                            // 0
		};
		write_sh_dx();
		write_sh_dy();
		write_sh_dz();
	}
}


template <typename scalar_t>
__global__ void kernel_sh_backward(
    const scalar_t * __restrict__ grad,
	const scalar_t * __restrict__ inputs,
    uint32_t B, uint32_t D, uint32_t C,
    const scalar_t * __restrict__ dy_dx,
    scalar_t * grad_inputs
) {
	const uint32_t t = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t b = t / D;
	if (b >= B) return;

	const uint32_t d = t - b * D;
	const uint32_t C2 = C * C;

	// locate
	grad += b * C2;
	dy_dx += b * D * C2 + d * C2;

	for (int ch = 0; ch < C2; ch++) {
		grad_inputs[t] += grad[ch] * dy_dx[ch];
		//printf("t=%d, b=%d, d=%d, ch=%d, grad=%f (+= %f * %f)\n", t, b, d, ch, grad_inputs[t], grad[ch], dy_dx[ch]);
	}

}

// inputs: [B, D], float, in [0, 1]
// outputs: [B, L * C], float
template <typename scalar_t>
void sh_encode_forward_cuda(const scalar_t *inputs, scalar_t *outputs, const uint32_t B, const uint32_t D, const uint32_t C, scalar_t *dy_dx) {
	static constexpr uint32_t N_THREADS = 256;
	kernel_sh<scalar_t><<<div_round_up(B, N_THREADS), N_THREADS>>>(inputs, outputs, B, D, C, dy_dx);
}


template <typename scalar_t>
void sh_encode_backward_cuda(const scalar_t *grad, const scalar_t *inputs, const uint32_t B, const uint32_t D, const uint32_t C, scalar_t *dy_dx, scalar_t *grad_inputs) {
	static constexpr uint32_t N_THREADS = 256;
	kernel_sh_backward<scalar_t><<<div_round_up(B * D, N_THREADS), N_THREADS>>>(grad, inputs, B, D, C, dy_dx, grad_inputs);
}


void sh_encode_forward(at::Tensor inputs, at::Tensor outputs, const uint32_t B, const uint32_t D, const uint32_t C, at::optional<at::Tensor> dy_dx) {
    CHECK_CUDA(inputs);
    CHECK_CUDA(outputs);
    // CHECK_CUDA(dy_dx);
    
    CHECK_CONTIGUOUS(inputs);
    CHECK_CONTIGUOUS(outputs);
    // CHECK_CONTIGUOUS(dy_dx);

    CHECK_IS_FLOATING(inputs);
    CHECK_IS_FLOATING(outputs);
    // CHECK_IS_FLOATING(dy_dx);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    inputs.scalar_type(), "sh_encode_forward_cuda", ([&] {
		sh_encode_forward_cuda<scalar_t>(inputs.data_ptr<scalar_t>(), outputs.data_ptr<scalar_t>(), B, D, C, dy_dx.has_value() ? dy_dx.value().data_ptr<scalar_t>() : nullptr);
    }));	
}

void sh_encode_backward(at::Tensor grad, at::Tensor inputs, const uint32_t B, const uint32_t D, const uint32_t C, at::Tensor dy_dx, at::Tensor grad_inputs) {    
    CHECK_CUDA(grad);
    CHECK_CUDA(inputs);
    CHECK_CUDA(dy_dx);
    CHECK_CUDA(grad_inputs);
    
    CHECK_CONTIGUOUS(grad);
    CHECK_CONTIGUOUS(inputs);
    CHECK_CONTIGUOUS(dy_dx);
    CHECK_CONTIGUOUS(grad_inputs);

    CHECK_IS_FLOATING(grad);
    CHECK_IS_FLOATING(inputs);
    CHECK_IS_FLOATING(dy_dx);
    CHECK_IS_FLOATING(grad_inputs);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad.scalar_type(), "sh_encode_backward_cuda", ([&] {
    	sh_encode_backward_cuda<scalar_t>(grad.data_ptr<scalar_t>(), inputs.data_ptr<scalar_t>(), B, D, C, dy_dx.data_ptr<scalar_t>(), grad_inputs.data_ptr<scalar_t>());
    }));	
}