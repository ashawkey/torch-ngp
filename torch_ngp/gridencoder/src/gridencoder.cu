#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <algorithm>
#include <stdexcept>

#include <stdint.h>
#include <cstdio>


#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")


// just for compatability of half precision in AT_DISPATCH_FLOATING_TYPES_AND_HALF...
static inline  __device__ at::Half atomicAdd(at::Half *address, at::Half val) {
  // requires CUDA >= 10 and ARCH >= 70
  // this is very slow compared to float or __half2, and never used.
  //return atomicAdd(reinterpret_cast<__half*>(address), val);
}


template <typename T>
static inline __host__ __device__ T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}


template <uint32_t D>
__device__ uint32_t fast_hash(const uint32_t pos_grid[D]) {
    static_assert(D <= 7, "fast_hash can only hash up to 7 dimensions.");

    // While 1 is technically not a good prime for hashing (or a prime at all), it helps memory coherence
    // and is sufficient for our use case of obtaining a uniformly colliding index from high-dimensional
    // coordinates.
    constexpr uint32_t primes[7] = { 1, 2654435761, 805459861, 3674653429, 2097192037, 1434869437, 2165219737 };

    uint32_t result = 0;
    #pragma unroll
    for (uint32_t i = 0; i < D; ++i) {
        result ^= pos_grid[i] * primes[i];
    }

    return result;
}


template <uint32_t D, uint32_t C>
__device__ uint32_t get_grid_index(const uint32_t gridtype, const bool align_corners, const uint32_t ch, const uint32_t hashmap_size, const uint32_t resolution, const uint32_t pos_grid[D]) {
    uint32_t stride = 1;
    uint32_t index = 0;

    #pragma unroll
    for (uint32_t d = 0; d < D && stride <= hashmap_size; d++) {
        index += pos_grid[d] * stride;
        stride *= align_corners ? resolution: (resolution + 1);
    }

    // NOTE: for NeRF, the hash is in fact not necessary. Check https://github.com/NVlabs/instant-ngp/issues/97.
    // gridtype: 0 == hash, 1 == tiled
    if (gridtype == 0 && stride > hashmap_size) {
        index = fast_hash<D>(pos_grid);
    }

    return (index % hashmap_size) * C + ch;
}


template <typename scalar_t, uint32_t D, uint32_t C>
__global__ void kernel_grid(
    const float * __restrict__ inputs, 
    const scalar_t * __restrict__ grid, 
    const int * __restrict__ offsets, 
    scalar_t * __restrict__ outputs, 
    const uint32_t B, const uint32_t L, const float S, const uint32_t H,
    scalar_t * __restrict__ dy_dx,
    const uint32_t gridtype,
    const bool align_corners
) {
    const uint32_t b = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (b >= B) return;

    const uint32_t level = blockIdx.y;
    
    // locate
    grid += (uint32_t)offsets[level] * C;
    inputs += b * D;
    outputs += level * B * C + b * C;

    // check input range (should be in [0, 1])
    bool flag_oob = false;
    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {
        if (inputs[d] < 0 || inputs[d] > 1) {
            flag_oob = true;
        }
    }
    // if input out of bound, just set output to 0
    if (flag_oob) {
        #pragma unroll
        for (uint32_t ch = 0; ch < C; ch++) {
            outputs[ch] = 0; 
        }
        if (dy_dx) {
            dy_dx += b * D * L * C + level * D * C; // B L D C
            #pragma unroll
            for (uint32_t d = 0; d < D; d++) {
                #pragma unroll
                for (uint32_t ch = 0; ch < C; ch++) {
                    dy_dx[d * C + ch] = 0; 
                }       
            }
        }
        return;
    }

    const uint32_t hashmap_size = offsets[level + 1] - offsets[level];
    const float scale = exp2f(level * S) * H - 1.0f;
    const uint32_t resolution = (uint32_t)ceil(scale) + 1;
    
    // calculate coordinate
    float pos[D];
    uint32_t pos_grid[D];

    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {
        pos[d] = inputs[d] * scale + (align_corners ? 0.0f : 0.5f);
        pos_grid[d] = floorf(pos[d]);
        pos[d] -= (float)pos_grid[d];
    }

    //printf("[b=%d, l=%d] pos=(%f, %f)+(%d, %d)\n", b, level, pos[0], pos[1], pos_grid[0], pos_grid[1]);

    // interpolate
    scalar_t results[C] = {0}; // temp results in register

    #pragma unroll
    for (uint32_t idx = 0; idx < (1 << D); idx++) {
        float w = 1;
        uint32_t pos_grid_local[D];

        #pragma unroll
        for (uint32_t d = 0; d < D; d++) {
            if ((idx & (1 << d)) == 0) {
                w *= 1 - pos[d];
                pos_grid_local[d] = pos_grid[d];
            } else {
                w *= pos[d];
                pos_grid_local[d] = pos_grid[d] + 1;
            }
        }

        uint32_t index = get_grid_index<D, C>(gridtype, align_corners, 0, hashmap_size, resolution, pos_grid_local);

        // writing to register (fast)
        #pragma unroll
        for (uint32_t ch = 0; ch < C; ch++) {
            results[ch] += w * grid[index + ch];
        }

        //printf("[b=%d, l=%d] int %d, idx %d, w %f, val %f\n", b, level, idx, index, w, grid[index]);
    }    

    // writing to global memory (slow)
    #pragma unroll
    for (uint32_t ch = 0; ch < C; ch++) {
        outputs[ch] = results[ch]; 
    }

    // prepare dy_dx
    // differentiable (soft) indexing: https://discuss.pytorch.org/t/differentiable-indexing/17647/9
    if (dy_dx) {

        dy_dx += b * D * L * C + level * D * C; // B L D C

        #pragma unroll
        for (uint32_t gd = 0; gd < D; gd++) {

            scalar_t results_grad[C] = {0};

            #pragma unroll
            for (uint32_t idx = 0; idx < (1 << (D - 1)); idx++) {
                float w = scale;
                uint32_t pos_grid_local[D];

                #pragma unroll
                for (uint32_t nd = 0; nd < D - 1; nd++) {
                    const uint32_t d = (nd >= gd) ? (nd + 1) : nd;

                    if ((idx & (1 << nd)) == 0) {
                        w *= 1 - pos[d];
                        pos_grid_local[d] = pos_grid[d];
                    } else {
                        w *= pos[d];
                        pos_grid_local[d] = pos_grid[d] + 1;
                    }
                }

                pos_grid_local[gd] = pos_grid[gd];
                uint32_t index_left = get_grid_index<D, C>(gridtype, align_corners, 0, hashmap_size, resolution, pos_grid_local);
                pos_grid_local[gd] = pos_grid[gd] + 1;
                uint32_t index_right = get_grid_index<D, C>(gridtype, align_corners, 0, hashmap_size, resolution, pos_grid_local);

                #pragma unroll
                for (uint32_t ch = 0; ch < C; ch++) {
                    results_grad[ch] += w * (grid[index_right + ch] - grid[index_left + ch]);
                }
            }

            #pragma unroll
            for (uint32_t ch = 0; ch < C; ch++) {
                dy_dx[gd * C + ch] = results_grad[ch];
            }
        }
    }
}


template <typename scalar_t, uint32_t D, uint32_t C, uint32_t N_C>
__global__ void kernel_grid_backward(
    const scalar_t * __restrict__ grad,
    const float * __restrict__ inputs, 
    const scalar_t * __restrict__ grid, 
    const int * __restrict__ offsets, 
    scalar_t * __restrict__ grad_grid, 
    const uint32_t B, const uint32_t L, const float S, const uint32_t H,
    const uint32_t gridtype,
    const bool align_corners
) {
    const uint32_t b = (blockIdx.x * blockDim.x + threadIdx.x) * N_C / C;
    if (b >= B) return;

    const uint32_t level = blockIdx.y;
    const uint32_t ch = (blockIdx.x * blockDim.x + threadIdx.x) * N_C - b * C;

    // locate
    grad_grid += offsets[level] * C;
    inputs += b * D;
    grad += level * B * C + b * C + ch; // L, B, C

    const uint32_t hashmap_size = offsets[level + 1] - offsets[level];
    const float scale = exp2f(level * S) * H - 1.0f;
    const uint32_t resolution = (uint32_t)ceil(scale) + 1;

    // check input range (should be in [0, 1])
    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {
        if (inputs[d] < 0 || inputs[d] > 1) {
            return; // grad is init as 0, so we simply return.
        }
    }

    // calculate coordinate
    float pos[D];
    uint32_t pos_grid[D];

    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {
        pos[d] = inputs[d] * scale + (align_corners ? 0.0f : 0.5f);
        pos_grid[d] = floorf(pos[d]);
        pos[d] -= (float)pos_grid[d];
    }

    scalar_t grad_cur[N_C] = {0}; // fetch to register
    #pragma unroll
    for (uint32_t c = 0; c < N_C; c++) {
        grad_cur[c] = grad[c];
    }

    // interpolate
    #pragma unroll
    for (uint32_t idx = 0; idx < (1 << D); idx++) {
        float w = 1;
        uint32_t pos_grid_local[D];

        #pragma unroll
        for (uint32_t d = 0; d < D; d++) {
            if ((idx & (1 << d)) == 0) {
                w *= 1 - pos[d];
                pos_grid_local[d] = pos_grid[d];
            } else {
                w *= pos[d];
                pos_grid_local[d] = pos_grid[d] + 1;
            }
        }

        uint32_t index = get_grid_index<D, C>(gridtype, align_corners, ch, hashmap_size, resolution, pos_grid_local);

        // atomicAdd for __half is slow (especially for large values), so we use __half2 if N_C % 2 == 0
        // TODO: use float which is better than __half, if N_C % 2 != 0
        if (std::is_same<scalar_t, at::Half>::value && N_C % 2 == 0) {
            #pragma unroll
            for (uint32_t c = 0; c < N_C; c += 2) {
                // process two __half at once (by interpreting as a __half2)
                __half2 v = {(__half)(w * grad_cur[c]), (__half)(w * grad_cur[c + 1])};
                atomicAdd((__half2*)&grad_grid[index + c], v);
            }
        // float, or __half when N_C % 2 != 0 (which means C == 1)
        } else {
            #pragma unroll
            for (uint32_t c = 0; c < N_C; c++) {
                atomicAdd(&grad_grid[index + c], w * grad_cur[c]);
            }
        }
    }    
}


template <typename scalar_t, uint32_t D, uint32_t C>
__global__ void kernel_input_backward(
    const scalar_t * __restrict__ grad,
    const scalar_t * __restrict__ dy_dx,  
    scalar_t * __restrict__ grad_inputs, 
    uint32_t B, uint32_t L
) {
    const uint32_t t = threadIdx.x + blockIdx.x * blockDim.x;
    if (t >= B * D) return;

    const uint32_t b = t / D;
    const uint32_t d = t - b * D;

    dy_dx += b * L * D * C;

    scalar_t result = 0;
    
    # pragma unroll
    for (int l = 0; l < L; l++) {
        # pragma unroll
        for (int ch = 0; ch < C; ch++) {
            result += grad[l * B * C + b * C + ch] * dy_dx[l * D * C + d * C + ch];
        }
    }

    grad_inputs[t] = result;
}


template <typename scalar_t, uint32_t D>
void kernel_grid_wrapper(const float *inputs, const scalar_t *embeddings, const int *offsets, scalar_t *outputs, const uint32_t B, const uint32_t C, const uint32_t L, const float S, const uint32_t H, scalar_t *dy_dx, const uint32_t gridtype, const bool align_corners) {
    static constexpr uint32_t N_THREAD = 512;
    const dim3 blocks_hashgrid = { div_round_up(B, N_THREAD), L, 1 };
    switch (C) {
        case 1: kernel_grid<scalar_t, D, 1><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, B, L, S, H, dy_dx, gridtype, align_corners); break;
        case 2: kernel_grid<scalar_t, D, 2><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, B, L, S, H, dy_dx, gridtype, align_corners); break;
        case 4: kernel_grid<scalar_t, D, 4><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, B, L, S, H, dy_dx, gridtype, align_corners); break;
        case 8: kernel_grid<scalar_t, D, 8><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, B, L, S, H, dy_dx, gridtype, align_corners); break;
        default: throw std::runtime_error{"GridEncoding: C must be 1, 2, 4, or 8."};
    }
}

// inputs: [B, D], float, in [0, 1]
// embeddings: [sO, C], float
// offsets: [L + 1], uint32_t
// outputs: [L, B, C], float (L first, so only one level of hashmap needs to fit into cache at a time.)
// H: base resolution
// dy_dx: [B, L * D * C]
template <typename scalar_t>
void grid_encode_forward_cuda(const float *inputs, const scalar_t *embeddings, const int *offsets, scalar_t *outputs, const uint32_t B, const uint32_t D, const uint32_t C, const uint32_t L, const float S, const uint32_t H, scalar_t *dy_dx, const uint32_t gridtype, const bool align_corners) {
    switch (D) {
        case 1: kernel_grid_wrapper<scalar_t, 1>(inputs, embeddings, offsets, outputs, B, C, L, S, H, dy_dx, gridtype, align_corners); break;
        case 2: kernel_grid_wrapper<scalar_t, 2>(inputs, embeddings, offsets, outputs, B, C, L, S, H, dy_dx, gridtype, align_corners); break;
        case 3: kernel_grid_wrapper<scalar_t, 3>(inputs, embeddings, offsets, outputs, B, C, L, S, H, dy_dx, gridtype, align_corners); break;
        case 4: kernel_grid_wrapper<scalar_t, 4>(inputs, embeddings, offsets, outputs, B, C, L, S, H, dy_dx, gridtype, align_corners); break;
        case 5: kernel_grid_wrapper<scalar_t, 5>(inputs, embeddings, offsets, outputs, B, C, L, S, H, dy_dx, gridtype, align_corners); break;
        default: throw std::runtime_error{"GridEncoding: D must be 1, 2, 3, 4, or 5."};
    }
    
}

template <typename scalar_t, uint32_t D>
void kernel_grid_backward_wrapper(const scalar_t *grad, const float *inputs, const scalar_t *embeddings, const int *offsets, scalar_t *grad_embeddings, const uint32_t B, const uint32_t C, const uint32_t L, const float S, const uint32_t H, scalar_t *dy_dx, scalar_t *grad_inputs, const uint32_t gridtype, const bool align_corners) {
    static constexpr uint32_t N_THREAD = 256;
    const uint32_t N_C = std::min(2u, C); // n_features_per_thread
    const dim3 blocks_hashgrid = { div_round_up(B * C / N_C, N_THREAD), L, 1 };
    switch (C) {
        case 1: 
            kernel_grid_backward<scalar_t, D, 1, 1><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, B, L, S, H, gridtype, align_corners); 
            if (dy_dx) kernel_input_backward<scalar_t, D, 1><<<div_round_up(B * D, N_THREAD), N_THREAD>>>(grad, dy_dx, grad_inputs, B, L);
            break;
        case 2: 
            kernel_grid_backward<scalar_t, D, 2, 2><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, B, L, S, H, gridtype, align_corners);
            if (dy_dx) kernel_input_backward<scalar_t, D, 2><<<div_round_up(B * D, N_THREAD), N_THREAD>>>(grad, dy_dx, grad_inputs, B, L);
            break;
        case 4: 
            kernel_grid_backward<scalar_t, D, 4, 2><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, B, L, S, H, gridtype, align_corners);
            if (dy_dx) kernel_input_backward<scalar_t, D, 4><<<div_round_up(B * D, N_THREAD), N_THREAD>>>(grad, dy_dx, grad_inputs, B, L);
            break;
        case 8: 
            kernel_grid_backward<scalar_t, D, 8, 2><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, B, L, S, H, gridtype, align_corners);
            if (dy_dx) kernel_input_backward<scalar_t, D, 8><<<div_round_up(B * D, N_THREAD), N_THREAD>>>(grad, dy_dx, grad_inputs, B, L);
            break;
        default: throw std::runtime_error{"GridEncoding: C must be 1, 2, 4, or 8."};
    }
}


// grad: [L, B, C], float
// inputs: [B, D], float, in [0, 1]
// embeddings: [sO, C], float
// offsets: [L + 1], uint32_t
// grad_embeddings: [sO, C]
// H: base resolution
template <typename scalar_t>
void grid_encode_backward_cuda(const scalar_t *grad, const float *inputs, const scalar_t *embeddings, const int *offsets, scalar_t *grad_embeddings, const uint32_t B, const uint32_t D, const uint32_t C, const uint32_t L, const float S, const uint32_t H, scalar_t *dy_dx, scalar_t *grad_inputs, const uint32_t gridtype, const bool align_corners) {
    switch (D) {
        case 1: kernel_grid_backward_wrapper<scalar_t, 1>(grad, inputs, embeddings, offsets, grad_embeddings, B, C, L, S, H, dy_dx, grad_inputs, gridtype, align_corners); break;
        case 2: kernel_grid_backward_wrapper<scalar_t, 2>(grad, inputs, embeddings, offsets, grad_embeddings, B, C, L, S, H, dy_dx, grad_inputs, gridtype, align_corners); break;
        case 3: kernel_grid_backward_wrapper<scalar_t, 3>(grad, inputs, embeddings, offsets, grad_embeddings, B, C, L, S, H, dy_dx, grad_inputs, gridtype, align_corners); break;
        case 4: kernel_grid_backward_wrapper<scalar_t, 4>(grad, inputs, embeddings, offsets, grad_embeddings, B, C, L, S, H, dy_dx, grad_inputs, gridtype, align_corners); break;
        case 5: kernel_grid_backward_wrapper<scalar_t, 5>(grad, inputs, embeddings, offsets, grad_embeddings, B, C, L, S, H, dy_dx, grad_inputs, gridtype, align_corners); break;
        default: throw std::runtime_error{"GridEncoding: D must be 1, 2, 3, 4, or 5."};
    }
}



void grid_encode_forward(const at::Tensor inputs, const at::Tensor embeddings, const at::Tensor offsets, at::Tensor outputs, const uint32_t B, const uint32_t D, const uint32_t C, const uint32_t L, const float S, const uint32_t H, at::optional<at::Tensor> dy_dx, const uint32_t gridtype, const bool align_corners) {
    CHECK_CUDA(inputs);
    CHECK_CUDA(embeddings);
    CHECK_CUDA(offsets);
    CHECK_CUDA(outputs);
    // CHECK_CUDA(dy_dx);
    
    CHECK_CONTIGUOUS(inputs);
    CHECK_CONTIGUOUS(embeddings);
    CHECK_CONTIGUOUS(offsets);
    CHECK_CONTIGUOUS(outputs);
    // CHECK_CONTIGUOUS(dy_dx);

    CHECK_IS_FLOATING(inputs);
    CHECK_IS_FLOATING(embeddings);
    CHECK_IS_INT(offsets);
    CHECK_IS_FLOATING(outputs);
    // CHECK_IS_FLOATING(dy_dx);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    embeddings.scalar_type(), "grid_encode_forward", ([&] {
        grid_encode_forward_cuda<scalar_t>(inputs.data_ptr<float>(), embeddings.data_ptr<scalar_t>(), offsets.data_ptr<int>(), outputs.data_ptr<scalar_t>(), B, D, C, L, S, H, dy_dx.has_value() ? dy_dx.value().data_ptr<scalar_t>() : nullptr, gridtype, align_corners);
    }));
}

void grid_encode_backward(const at::Tensor grad, const at::Tensor inputs, const at::Tensor embeddings, const at::Tensor offsets, at::Tensor grad_embeddings, const uint32_t B, const uint32_t D, const uint32_t C, const uint32_t L, const float S, const uint32_t H, const at::optional<at::Tensor> dy_dx, at::optional<at::Tensor> grad_inputs, const uint32_t gridtype, const bool align_corners) {
    CHECK_CUDA(grad);
    CHECK_CUDA(inputs);
    CHECK_CUDA(embeddings);
    CHECK_CUDA(offsets);
    CHECK_CUDA(grad_embeddings);
    // CHECK_CUDA(dy_dx);
    // CHECK_CUDA(grad_inputs);
    
    CHECK_CONTIGUOUS(grad);
    CHECK_CONTIGUOUS(inputs);
    CHECK_CONTIGUOUS(embeddings);
    CHECK_CONTIGUOUS(offsets);
    CHECK_CONTIGUOUS(grad_embeddings);
    // CHECK_CONTIGUOUS(dy_dx);
    // CHECK_CONTIGUOUS(grad_inputs);

    CHECK_IS_FLOATING(grad);
    CHECK_IS_FLOATING(inputs);
    CHECK_IS_FLOATING(embeddings);
    CHECK_IS_INT(offsets);
    CHECK_IS_FLOATING(grad_embeddings);
    // CHECK_IS_FLOATING(dy_dx);
    // CHECK_IS_FLOATING(grad_inputs);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad.scalar_type(), "grid_encode_backward", ([&] {
        grid_encode_backward_cuda<scalar_t>(grad.data_ptr<scalar_t>(), inputs.data_ptr<float>(), embeddings.data_ptr<scalar_t>(), offsets.data_ptr<int>(), grad_embeddings.data_ptr<scalar_t>(), B, D, C, L, S, H, dy_dx.has_value() ? dy_dx.value().data_ptr<scalar_t>() : nullptr, grad_inputs.has_value() ? grad_inputs.value().data_ptr<scalar_t>() : nullptr, gridtype, align_corners);
    }));
    
}
