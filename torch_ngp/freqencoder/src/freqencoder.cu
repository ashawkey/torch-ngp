#include "hip/hip_runtime.h"
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <algorithm>
#include <stdexcept>

#include <cstdio>


#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")

inline constexpr __device__ float PI() { return 3.141592653589793f; }

template <typename T>
__host__ __device__ T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}

// inputs: [B, D]
// outputs: [B, C], C = D + D * deg * 2
__global__ void kernel_freq(
    const float * __restrict__ inputs, 
    uint32_t B, uint32_t D, uint32_t deg, uint32_t C,
    float * outputs
) {
    // parallel on per-element
    const uint32_t t = threadIdx.x + blockIdx.x * blockDim.x;
    if (t >= B * C) return;

    // get index
    const uint32_t b = t / C;
    const uint32_t c = t - b * C; // t % C;

    // locate
    inputs += b * D;
    outputs += t;

    // write self
    if (c < D) {
        outputs[0] = inputs[c];
    // write freq
    } else {
        const uint32_t col = c / D - 1;
        const uint32_t d = c % D;
        const uint32_t freq = col / 2;
        const float phase_shift = (col % 2) * (PI() / 2);
        outputs[0] = __sinf(scalbnf(inputs[d], freq) + phase_shift);
    }
}

// grad: [B, C], C = D + D * deg * 2
// outputs: [B, C]
// grad_inputs: [B, D]
__global__ void kernel_freq_backward(
    const float * __restrict__ grad,
    const float * __restrict__ outputs,
    uint32_t B, uint32_t D, uint32_t deg, uint32_t C,
    float * grad_inputs
) {
    // parallel on per-element
    const uint32_t t = threadIdx.x + blockIdx.x * blockDim.x;
    if (t >= B * D) return;

    const uint32_t b = t / D;
    const uint32_t d = t - b * D; // t % D;

    // locate
    grad += b * C;
    outputs += b * C;
    grad_inputs += t;

    // register 
    float result = grad[d];
    grad += D;
    outputs += D;

    for (uint32_t f = 0; f < deg; f++) {
        result += scalbnf(1.0f, f) * (grad[d] * outputs[D + d] - grad[D + d] * outputs[d]);
        grad += 2 * D;
        outputs += 2 * D;
    }

    // write
    grad_inputs[0] = result;
}


void freq_encode_forward(at::Tensor inputs, const uint32_t B, const uint32_t D, const uint32_t deg, const uint32_t C, at::Tensor outputs) {
    CHECK_CUDA(inputs);
    CHECK_CUDA(outputs);
    
    CHECK_CONTIGUOUS(inputs);
    CHECK_CONTIGUOUS(outputs);

    CHECK_IS_FLOATING(inputs);
    CHECK_IS_FLOATING(outputs);

    static constexpr uint32_t N_THREADS = 128;

    kernel_freq<<<div_round_up(B * C, N_THREADS), N_THREADS>>>(inputs.data_ptr<float>(), B, D, deg, C, outputs.data_ptr<float>());
}


void freq_encode_backward(at::Tensor grad, at::Tensor outputs, const uint32_t B, const uint32_t D, const uint32_t deg, const uint32_t C, at::Tensor grad_inputs) {
    CHECK_CUDA(grad);
    CHECK_CUDA(outputs);
    CHECK_CUDA(grad_inputs);
    
    CHECK_CONTIGUOUS(grad);
    CHECK_CONTIGUOUS(outputs);
    CHECK_CONTIGUOUS(grad_inputs);

    CHECK_IS_FLOATING(grad);
    CHECK_IS_FLOATING(outputs);
    CHECK_IS_FLOATING(grad_inputs);

    static constexpr uint32_t N_THREADS = 128;

    kernel_freq_backward<<<div_round_up(B * D, N_THREADS), N_THREADS>>>(grad.data_ptr<float>(), outputs.data_ptr<float>(), B, D, deg, C, grad_inputs.data_ptr<float>());
}